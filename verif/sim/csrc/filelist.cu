PIC_LD=ld

ARCHIVE_OBJS=
ARCHIVE_OBJS += _14180_archive_1.so
_14180_archive_1.so : archive.0/_14180_archive_1.a
	@$(AR) -s $<
	@$(PIC_LD) -shared  -Bsymbolic  -o .//../simv.daidir//_14180_archive_1.so --whole-archive $< --no-whole-archive
	@rm -f $@
	@ln -sf .//../simv.daidir//_14180_archive_1.so $@





O0_OBJS =

$(O0_OBJS) : %.o: %.c
	$(CC_CG) $(CFLAGS_O0) -c -o $@ $<
 

%.o: %.c
	$(CC_CG) $(CFLAGS_CG) -c -o $@ $<
CU_UDP_OBJS = \


CU_LVL_OBJS = \
SIM_l.o 

MAIN_OBJS = \
objs/amcQw_d.o 

CU_OBJS = $(MAIN_OBJS) $(ARCHIVE_OBJS) $(CU_UDP_OBJS) $(CU_LVL_OBJS)

